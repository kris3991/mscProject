#include "hip/hip_runtime.h"
#include "../include/rxMesh.cuh"

//random uniform dist
//this code to select random point is referred from stack overflow.
//random selection for seed elements 
template<typename Iter, typename RandomGenerator>
Iter select_randomly(Iter start, Iter end, RandomGenerator& g) {
	std::uniform_int_distribution<> dis(0, std::distance(start, end) - 1);
	std::advance(start, dis(g));
	return start;
}

template<typename Iter>
Iter select_randomly(Iter start, Iter end) {
	static std::random_device rd;
	static std::mt19937 gen(rd());
	return select_randomly(start, end, gen);
}

preRxMeshDataStructure* preRxMeshDataStructure::rxMeshStruct = nullptr;


preRxMeshDataStructure* preRxMeshDataStructure::GetInstance()
{
	if (rxMeshStruct == nullptr) {
		rxMeshStruct = new preRxMeshDataStructure();
	}
	return rxMeshStruct;
}




preRxMeshDataStructure::preRxMeshDataStructure()
{
	d_faceVector = 0;
	d_adjascentTriangles = 0;
	h_adjascentTriangles.clear();
	sizeofFaceVector = 0;
	patchSize = 0;
	patchCount = 0;
	numFaces = 0;
}

void preRxMeshDataStructure::freeCudaData()
{
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if(d_faceVector != nullptr)
		hipFree(d_faceVector);
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if (d_sizeN != nullptr)
		hipFree(d_sizeN);
	if (d_patchingArray != nullptr)
		hipFree(d_patchingArray);
	if (d_boundaryElements != nullptr)
		hipFree(d_boundaryElements);
}

preRxMeshDataStructure::~preRxMeshDataStructure()
{

}

void preRxMeshDataStructure::initialise(TriangleMesh* tm)
{
	//in a manifold max number of adjascent triangles is 3. so adjascent 
	//tm->faceVector contains all the vertices in a face, dont confuse with h_faceIndexVector
	numFaces = tm->faceVector.size() / 3;
	int size_N = tm->faceVector.size();
	h_faceIndexVector.resize(numFaces);
	h_patchingArray.resize(numFaces);
	std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);



	h_boundaryElements.resize(numFaces);
	std::fill(h_boundaryElements.begin(), h_boundaryElements.end(), 0);

	for (int i = 0; i < numFaces; ++i)
	{
		h_faceIndexVector[i] = i;
	}
	sizeofFaceVector = sizeof(int) * tm->faceVector.size();

	//initialise cuda data
	hipError_t status;
	status = hipMalloc(&d_adjascentTriangles, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_adjascentTriangles" << std::endl;
	}
	status = hipMalloc(&d_faceVector, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_faceVector" << std::endl;
	}

	//allocate patching array.
	status = hipMalloc(&d_patchingArray, sizeof(int) * numFaces);
	if (status != hipSuccess)
	{
		std::cout << "Patching allocation failed" << std::endl;
	}

	status = hipMalloc(&d_boundaryElements, sizeof(int) * numFaces);
	if (status != hipSuccess)
	{
		std::cout << "Patching allocation failed" << std::endl;
	}


	//memcpy
	status = hipMemcpy(d_faceVector, tm->faceVector.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_faceVector" << std::endl;
	}


	//this is for cudaAtomic operations.

	std::vector<int> temp(tm->faceVector.size(), -1);
	status = hipMemcpy(d_adjascentTriangles, temp.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_adjascentTriangles" << std::endl;
	}
}

void preRxMeshDataStructure::h_initialiseSeedElements(TriangleMesh* tm, ComponentManager* cm, int pc)
{
	//if you create a patch bigger than the face count. the mesh will probably get messed up during patching.
	
	
	patchCount = pc;
	//each face has 3 elements]
	//last patch will have less elements depending on the patchCount;
	
	patchSize = (numFaces + patchCount - 1) / patchCount;
	int temp{ 0 };
	
	if (cm->componentCount == 1)
	{
		//o(n2) but n is small.
		
		int count{ 0 };
		int i = { 1 };
		int begin{ 0 }, end{ 0 };
		int temp{ 0 };
		while (count < patchCount)
		{
			end += patchSize;
			if (end > numFaces)
				end = numFaces;

			int t0{-1}, t1{ -1 }, t2{ -1 };
			while (t0 == -1 && t1 == -1 && t2 == -1)
			{
				temp = *select_randomly(h_faceIndexVector.begin() + begin, h_faceIndexVector.begin() + end - 1);
				t0 = h_adjTriMap[temp][0];
				t1 = h_adjTriMap[temp][2];
				t2 = h_adjTriMap[temp][2];
			}
			h_seedElements.push_back(temp);
			begin += patchSize;

			count++;
		}
			
	}
	//allocate seed array.
	hipError_t status = hipMalloc(&d_seedArray, sizeof(int) * h_seedElements.size());
	if (status != hipSuccess)
	{
		std::cout << "allocation error for d_seedArray" << std::endl;
	}
	hipMemcpy(d_seedArray, h_seedElements.data(), sizeof(int)* h_seedElements.size(), hipMemcpyHostToDevice);
}

void preRxMeshDataStructure::h_initialiseSeedElementsMultiComp(TriangleMesh* tm, ComponentManager* cm)
{
	multiComponentPatchSize.clear();
	multiComponentPatchSize.resize(cm->componentCount);
	
	for (int i = 0; i < cm->componentCount; ++i)
	{
		int count{ 0 };
		int begin = cm->componentLocation[i], end = cm->componentLocation[i];
		int temp{ 0 };
		int start = cm->componentLocation[i];
		int stop = cm->componentLocation[i + 1];
		int faceCount = stop - start;
		multiComponentPatchSize[i] = (faceCount + multiComponentPatchCount[i] - 1) / multiComponentPatchCount[i];
		while (count < multiComponentPatchCount[i])
		{
			end += multiComponentPatchSize[i];
			if (end > numFaces)
				end = numFaces;

			int t0{ -1 }, t1{ -1 }, t2{-1};
			while (t0 == -1 && t1 == -1 && t2 == -1)
			{
				temp = *select_randomly(h_faceIndexVector.begin() + begin, h_faceIndexVector.begin() + end - 1);
				t0 = h_adjTriMap[temp][0];
				t1 = h_adjTriMap[temp][2];
				t2 = h_adjTriMap[temp][2];
			}
			h_seedElements.push_back(temp);
			begin += multiComponentPatchSize[i];

			count++;
		}
	}
}

void preRxMeshDataStructure::clear()
{
	h_seedElements.clear();
	h_adjascentTriangles.clear();
	multiComponentPatchCount.clear();
	multiComponentPatchSize.clear();
	h_adjTriMap.clear();
}

void preRxMeshDataStructure::clearSeedComponents(TriangleMesh* tm)
{
	h_seedElements.clear();
	multiComponentPatchSize.clear();
	int size_N = tm->faceVector.size() / 3;
	h_patchingArray.resize(size_N);
	std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);
}


void preRxMeshDataStructure::h_fillAdjascentTriangles(TriangleMesh* tm)
{
	clear();
	int size_N = tm->faceVector.size();
	int threadCount = 1 << 10;
	if (threadCount > size_N)
		threadCount = size_N;

	int gridSize = (size_N + threadCount - 1) / threadCount;
	h_adjascentTriangles.resize(size_N);
	int sharedMemorySize = 2 * threadCount;
	//in a manifold the max number of faces adjascent to one face is 3, but boundary vertices have -1 in this implementation.
	d_fillAdjascentTriangles << <gridSize, threadCount >> > (d_faceVector, d_adjascentTriangles, size_N);
	//copy the data for later operations.
	hipMemcpy(h_adjascentTriangles.data(), d_adjascentTriangles, sizeofFaceVector, hipMemcpyDeviceToHost);

	//fill hash map for faster access.
	for (int i = 0; i < h_adjascentTriangles.size(); i = i + 3)
	{
		h_adjTriMap[i / 3] = { h_adjascentTriangles[i], h_adjascentTriangles[i + 1], h_adjascentTriangles[i + 2] };
	}
}

__global__
void d_fillAdjascentTriangles(int* d_faceVector, int* d_adjascentTriangles, int size_N)
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	int lId = threadIdx.x;
	//basic modulo operation for triangles.

	if (tId < size_N)
	{
		int v0 = d_faceVector[tId];
		int v1 = 0;
		if (tId % 3 == 2)
			v1 = d_faceVector[tId - 2];
		else
			v1 = d_faceVector[tId + 1];
		int v2, v3;
		for (int i = 0; i < size_N; ++i)
		{
			v2 = d_faceVector[i];
			if(i % 3 == 2)
				v3 = d_faceVector[i - 2];
			else
				v3 = d_faceVector[i + 1];
			if (v0 == v3 && v1 == v2)
			{
				d_adjascentTriangles[tId] = i/3;
			}
		}

	}
}

__global__
void d_populatePatchingArray(int* d_patchingArray, int size_N, int* d_adjascentTriangles, int* d_boundaryElements)
{
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	if (tId < size_N)
	{
		//the idea is to check for the faces who have adjascent elements in a different patch.
		//store that in boundary.
		//populate based on adj triangles. so no invalid triangle pops up in the patch.
		//thread divergence better than o(n3)
		int begin = blockIdx.x * blockDim.x;
		int end = blockIdx.x* blockDim.x + blockDim.x;

		int t0, t1, t2;
		if (d_patchingArray[tId] != -1)
		{
			int patch = tId;
			t0 = d_adjascentTriangles[3 * tId];
			if (t0 > end || t0 < begin)
			{
				atomicCAS(d_boundaryElements + patch, 0, 1);
			}

			t1 = d_adjascentTriangles[3 * tId + 1];

			if (t1 > end || t1 < begin)
			{
				atomicCAS(d_boundaryElements + patch, 0, 1);
			}


			t2 = d_adjascentTriangles[3 * tId + 2];

			if (t2 > end || t2 < begin)
			{
				atomicCAS(d_boundaryElements + patch, 0, 1);
			}


			if (t0 != -1 && atomicCAS(d_patchingArray + t0, -1, blockIdx.x))
			{
				
			}
			if (t1 != -1 && atomicCAS(d_patchingArray + t1, -1, blockIdx.x))
			{
				
			}
			if (t2 != -1 && atomicCAS(d_patchingArray + t2, -1, blockIdx.x))
			{
				
			}
		}
	}
}

//__global__ void d_populatePatchingArray(int* d_patchingArray, int size_N, int* d_adjascentTriangles, int* d_count, bool* d_continue) {
//	int tId = blockIdx.x * blockDim.x + threadIdx.x;
//
//	while (*d_continue) {
//		if (tId < size_N) {
//			int t0, t1, t2;
//
//			if (d_patchingArray[tId] != -1) 
//			{
//				t0 = d_adjascentTriangles[3 * tId];
//				t1 = d_adjascentTriangles[3 * tId + 1];
//				t2 = d_adjascentTriangles[3 * tId + 2];
//
//				if (t0 != -1 && d_patchingArray[t0] == -1) {
//					d_patchingArray[t0] = t0;
//					*d_continue = true;
//				}
//				if (t1 != -1 && d_patchingArray[t1] == -1) {
//					d_patchingArray[t1] = t1;
//					*d_continue = true;
//				}
//				if (t2 != -1 && d_patchingArray[t2] == -1) {
//					d_patchingArray[t2] = t2;
//					*d_continue = true;
//				}
//			}
//		}
//		__syncthreads();
//
//		if (tId == 0) {
//			*d_continue = false;
//		}
//		__syncthreads();
//	}
//}



__global__
void d_counter(int* d_patchingArray, int size_N, int* d_count)
{
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	*d_count = 0;
	if (tId < size_N)
	{
		if (d_patchingArray[tId] == -1)
			atomicAdd(d_count, 1);
	}
}

void preRxMeshDataStructure::h_fillPatchingArrayWithSeedPoints()
{
	//there is no point in parallelising this block.
	//and will only be done once.
	for (int i = 0; i < h_seedElements.size(); ++i)
	{
		int currFace = h_seedElements[i];
		h_patchingArray[currFace] = i;
	}

	hipError_t status = hipMemcpy(d_patchingArray, h_patchingArray.data(), sizeof(int) * h_patchingArray.size(), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_patchingArray" << std::endl;
	}

	//fill 0 for boundary just in case.
	status = hipMemcpy(d_boundaryElements, h_boundaryElements.data(), sizeof(int) * h_patchingArray.size(), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_patchingArray" << std::endl;
	}

}

void preRxMeshDataStructure::h_populatePatches(TriangleMesh* tm, bool doIterations, ComponentManager* cm, int pc)
{

	//the algorithm involvest the following steps
	//initialise seed elements.
	//copy seed elements to patching array
	//for every non -1 element in patching array, add its neighbours.
	//keep counter to keep track of the patching process.
	//check if the faces are boundary. Select non boundary faces as seed for next itertaion.
	//repeat until 5th loop if iteration is enabled. 

	
	std::random_device rd;
	std::mt19937 gen(rd());


	clearSeedComponents(tm);
	h_initialiseSeedElements(tm, cm, pc);

	// i am putting 5 loops as convergence max.
	
	int loopCounter = 5;
	
	do{
		h_fillPatchingArrayWithSeedPoints();
		//clear the gpu values 
		hipMemset(d_patchingArray, -1, tm->faceVector.size()/3);
		hipMemset(d_boundaryElements, 0, tm->faceVector.size() / 3);

		int threadCount = patchSize;
		int blockCount = patchCount;
		int size_N = tm->faceVector.size() / 3;
		int sharedMemorySize = threadCount * sizeof(int);
		//set any random non zero value.
		int count = 0;
		int* d_count = 0;
		hipMalloc(&d_count, sizeof(int));
		hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
		//i was using blelloch earlier to get the sum of all face values.
		//buts its easier to check for the number of -1s in the patching array
		/*for (int i = 0; i < 3; ++i)*/
		do
		{
			hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
			d_populatePatchingArray << <blockCount, threadCount >> > (d_patchingArray, size_N, d_adjascentTriangles, d_boundaryElements);
			hipDeviceSynchronize();
			d_counter << <blockCount, threadCount >> > (d_patchingArray, size_N, d_count);
			hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
		} while (count != 0);

		
		hipMemcpy(h_boundaryElements.data(), d_boundaryElements, sizeof(int) * size_N, hipMemcpyDeviceToHost);

		//in the next iteration choose a seed point that is not a boundary element.
		//since i have chosen seed points which are not really boundary the following step could be possibly be skipped for more performance.
		//adding a condition to skip this.
		h_seedElements.clear();
		if (doIterations)
		{
			//nothing to parallelise here.
			for (int i = 0; i < patchCount; ++i)
			{
				int random_number;
				int begin = i * patchSize;
				int end = (i + 1) * patchSize;
				std::uniform_int_distribution<> distr(begin, end);
				int val = 1;
				//this loop will almost always be 1,for example in a patch(size 180) of a sphere obj of 960 faces, 10 are boundary around 0.02 percent in each patch.
				//while worst case is o(n3) here this will generally be o(1).
				while (val)
				{
					random_number = distr(gen);
					val = h_boundaryElements[random_number];
				}
				h_seedElements.push_back(random_number);
			}
			loopCounter--;
		}
		else
		{
			loopCounter = 0;
		}
		hipFree(d_count);
		//clear all the cpu arrays.
		std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);
		std::fill(h_boundaryElements.begin(), h_boundaryElements.end(), 0);
	


	}while (loopCounter);

}



