#include "hip/hip_runtime.h"
#include "../include/rxMesh.cuh"

//random uniform dist
//this code to select random point is referred from stack overflow.
//random selection for seed elements 
template<typename Iter, typename RandomGenerator>
Iter select_randomly(Iter start, Iter end, RandomGenerator& g) {
	std::uniform_int_distribution<> dis(0, std::distance(start, end) - 1);
	std::advance(start, dis(g));
	return start;
}

template<typename Iter>
Iter select_randomly(Iter start, Iter end) {
	static std::random_device rd;
	static std::mt19937 gen(rd());
	return select_randomly(start, end, gen);
}

preRxMeshDataStructure* preRxMeshDataStructure::rxMeshStruct = nullptr;


preRxMeshDataStructure* preRxMeshDataStructure::GetInstance()
{
	if (rxMeshStruct == nullptr) {
		rxMeshStruct = new preRxMeshDataStructure();
	}
	return rxMeshStruct;
}




preRxMeshDataStructure::preRxMeshDataStructure()
{
	d_faceVector = 0;
	d_adjascentTriangles = 0;
	h_adjascentTriangles.clear();
	sizeofFaceVector = 0;
	patchSize = 0;
	patchCount = 0;
	numFaces = 0;
}

void preRxMeshDataStructure::freeCudaData()
{
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if(d_faceVector != nullptr)
		hipFree(d_faceVector);
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if (d_sizeN != nullptr)
		hipFree(d_sizeN);
}

preRxMeshDataStructure::~preRxMeshDataStructure()
{

}

void preRxMeshDataStructure::initialise(TriangleMesh* tm)
{
	//in a manifold max number of adjascent triangles is 3. so adjascent 
	//tm->faceVector contains all the vertices in a face, dont confuse with h_faceIndexVector
	numFaces = tm->faceVector.size() / 3;
	int size_N = tm->faceVector.size();
	h_faceIndexVector.resize(numFaces);
	for (int i = 0; i < numFaces; ++i)
	{
		h_faceIndexVector[i] = i;
	}
	sizeofFaceVector = sizeof(int) * tm->faceVector.size();

	//initialise cuda data
	hipError_t status;
	status = hipMalloc(&d_adjascentTriangles, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_adjascentTriangles" << std::endl;
	}
	status = hipMalloc(&d_faceVector, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_faceVector" << std::endl;
	}
	status = hipMalloc(&d_adjascentTriangles, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_adjascentTriangles" << std::endl;
	}
	//end

	//memcpy
	status = hipMemcpy(d_faceVector, tm->faceVector.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_faceVector" << std::endl;
	}

	//this is for cudaAtomic operations.

	std::vector<int> temp(tm->faceVector.size(), -1);
	status = hipMemcpy(d_adjascentTriangles, temp.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_adjascentTriangles" << std::endl;
	}
}

void preRxMeshDataStructure::h_initialiseSeedElements(TriangleMesh* tm, ComponentManager* cm, int ps)
{
	//if you create a patch bigger than the face count. the mesh will probably get messed up during patching.
	
	
	patchSize = ps;
	//each face has 3 elements
	
	patchCount = (numFaces + patchSize - 1) / patchSize;
	int temp{ 0 };
	
	if (cm->componentCount == 1)
	{
		//o(n2) but n is small.
		for (int i = 0; i < patchCount; ++i)
		{
			
			//this is for faster convergenece.
			//select seed points that have 3 adjascent edges, i.e. not boundary.
			//otherwise patching algorithm will not work.
			int next{ -1 }, previous{ -1 };
			while (next == -1 && next == -1)
			{
				temp = *select_randomly(h_faceIndexVector.begin(), h_faceIndexVector.end());
				if (temp % 3 == 2)
				{
					next = h_adjascentTriangles[temp - 2];
					previous = h_adjascentTriangles[temp - 1];
				}
				else if (temp % 3 == 1)
				{
					next = h_adjascentTriangles[temp + 1];
					previous = h_adjascentTriangles[temp - 1];
				}
				else
				{
					next = h_adjascentTriangles[temp + 1];
					previous = h_adjascentTriangles[temp + 2];
				}
			}
			h_seedElements.push_back(temp);

		}
			
	}
	else
	{
		//from component manager we basically have idea on where a component begins and ends.
		//the idea is to split face data into patching data based on component size.
		int totalPatchSpace = patchCount * patchSize;
		
		

			int count{ 0 };
			int i = { 1 };
			int begin{ 0 }, end{ 0 };
			int temp{ 0 };
			while (count <  patchCount)
			{
				int offset = (count + 1) * patchSize;
				
				//if for some reason you entered a patch size > component patch count, that needs to be handled
				if (offset > cm->componentLocation[i] - cm->componentLocation[0] && offset < totalPatchSpace)
				{
					i++;
				}
				
				begin = cm->componentLocation[i - 1];
				end = cm->componentLocation[i];
				int next{ -1 }, previous{ -1 };
				while (next == -1 && next == -1)
				{
					temp = *select_randomly(h_faceIndexVector.begin() + begin, h_faceIndexVector.begin() + end);
					if (temp % 3 == 2)
					{
						next = h_adjascentTriangles[temp - 2];
						previous = h_adjascentTriangles[temp - 1];
					}
					else if (temp % 3 == 1)
					{
						next = h_adjascentTriangles[temp + 1];
						previous = h_adjascentTriangles[temp - 1];
					}
					else
					{
						next = h_adjascentTriangles[temp + 1];
						previous = h_adjascentTriangles[temp + 2];
					}
				}
				h_seedElements.push_back(temp);
				count++;
			}

	}

}

void preRxMeshDataStructure::clear()
{
	h_seedElements.clear();
	h_adjascentTriangles.clear();
}


void preRxMeshDataStructure::h_fillAdjascentTriangles(TriangleMesh* tm)
{
	clear();
	int size_N = tm->faceVector.size();
	int threadCount = 1 << 10;
	if (threadCount > size_N)
		threadCount = size_N;

	int gridSize = (size_N + threadCount - 1) / threadCount;
	h_adjascentTriangles.resize(size_N);
	int sharedMemorySize = 2 * threadCount;
	//in a manifold the max number of faces adjascent to one face is 3, but boundary vertices have -1 in this implementation.
	d_fillAdjascentTriangles << <gridSize, threadCount >> > (d_faceVector, d_adjascentTriangles, size_N);
	//copy the data for later operations.
	hipMemcpy(h_adjascentTriangles.data(), d_adjascentTriangles, sizeofFaceVector, hipMemcpyDeviceToHost);

}

__global__
void d_fillAdjascentTriangles(int* d_faceVector, int* d_adjascentTriangles, int size_N)
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	int lId = threadIdx.x;
	
	if (tId < size_N)
	{
		int v0 = d_faceVector[tId];
		int v1 = 0;
		if (tId % 3 == 2)
			v1 = d_faceVector[tId - 2];
		else
			v1 = d_faceVector[tId + 1];
		int v2, v3;
		for (int i = 0; i < size_N; ++i)
		{
			v2 = d_faceVector[i];
			if(i % 3 == 2)
				v3 = d_faceVector[i - 2];
			else
				v3 = d_faceVector[i + 1];
			if (v0 == v3 && v1 == v2)
			{
				d_adjascentTriangles[tId] = i/3;
			}
		}

	}
}