#include "hip/hip_runtime.h"
#include "../include/rxMesh.cuh"

//random uniform dist
//this code to select random point is referred from stack overflow.
//random selection for seed elements 
template<typename Iter, typename RandomGenerator>
Iter select_randomly(Iter start, Iter end, RandomGenerator& g) {
	std::uniform_int_distribution<> dis(0, std::distance(start, end) - 1);
	std::advance(start, dis(g));
	return start;
}

template<typename Iter>
Iter select_randomly(Iter start, Iter end) {
	static std::random_device rd;
	static std::mt19937 gen(rd());
	return select_randomly(start, end, gen);
}

preRxMeshDataStructure* preRxMeshDataStructure::rxMeshStruct = nullptr;


preRxMeshDataStructure* preRxMeshDataStructure::GetInstance()
{
	if (rxMeshStruct == nullptr) {
		rxMeshStruct = new preRxMeshDataStructure();
	}
	return rxMeshStruct;
}




preRxMeshDataStructure::preRxMeshDataStructure()
{
	d_faceVector = 0;
	d_adjascentTriangles = 0;
	h_adjascentTriangles.clear();
	sizeofFaceVector = 0;
	patchSize = 0;
	patchCount = 0;
	numFaces = 0;
}

void preRxMeshDataStructure::freeCudaData()
{
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if(d_faceVector != nullptr)
		hipFree(d_faceVector);
	if(d_adjascentTriangles != nullptr)
		hipFree(d_adjascentTriangles);
	if (d_sizeN != nullptr)
		hipFree(d_sizeN);
	if (d_patchingArray != nullptr)
		hipFree(d_patchingArray);
	if (d_boundaryElements != nullptr)
		hipFree(d_boundaryElements);
	if (d_patchPositions != nullptr)
		hipFree(d_patchPositions);
}

preRxMeshDataStructure::~preRxMeshDataStructure()
{

}

void preRxMeshDataStructure::initialise(TriangleMesh* tm)
{
	//in a manifold max number of adjascent triangles is 3. so adjascent 
	//tm->faceVector contains all the vertices in a face, dont confuse with h_faceIndexVector
	numFaces = tm->faceVector.size() / 3;
	int size_N = tm->faceVector.size();
	h_faceIndexVector.resize(numFaces);
	h_patchingArray.resize(numFaces);
	std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);



	h_boundaryElements.resize(numFaces);
	std::fill(h_boundaryElements.begin(), h_boundaryElements.end(), 0);

	for (int i = 0; i < numFaces; ++i)
	{
		h_faceIndexVector[i] = i;
	}
	sizeofFaceVector = sizeof(int) * tm->faceVector.size();

	//initialise cuda data
	hipError_t status;
	status = hipMalloc(&d_adjascentTriangles, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_adjascentTriangles" << std::endl;
	}
	status = hipMalloc(&d_faceVector, sizeofFaceVector);
	if (status != hipSuccess)
	{
		std::cout << "error allocating d_faceVector" << std::endl;
	}

	//allocate patching array.
	status = hipMalloc(&d_patchingArray, sizeof(int) * numFaces);
	if (status != hipSuccess)
	{
		std::cout << "Patching allocation failed" << std::endl;
	}

	status = hipMalloc(&d_boundaryElements, sizeof(int) * numFaces);
	if (status != hipSuccess)
	{
		std::cout << "Patching allocation failed" << std::endl;
	}


	//memcpy
	status = hipMemcpy(d_faceVector, tm->faceVector.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_faceVector" << std::endl;
	}


	//this is for cudaAtomic operations.

	std::vector<int> temp(tm->faceVector.size(), -1);
	status = hipMemcpy(d_adjascentTriangles, temp.data(), sizeofFaceVector, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_adjascentTriangles" << std::endl;
	}
}

void preRxMeshDataStructure::h_initialiseSeedElements(TriangleMesh* tm, ComponentManager* cm, int pc)
{
	//if you create a patch bigger than the face count. the mesh will probably get messed up during patching.
	
	
	patchCount = pc;
	//each face has 3 elements]
	//last patch will have less elements depending on the patchCount;
	
	patchSize = (numFaces + patchCount - 1) / patchCount;
	int temp{ 0 };
	
	if (cm->componentCount == 1)
	{
		//o(n2) but n is small.
		
		int count{ 0 };
		int i = { 1 };
		int begin{ 0 }, end{ 0 };
		int temp{ 0 };
		while (count < patchCount)
		{
			end += patchSize;
			if (end > numFaces)
				end = numFaces;

			int t0{-1}, t1{ -1 }, t2{ -1 };
			while (t0 == -1 && t1 == -1 && t2 == -1)
			{
				temp = *select_randomly(h_faceIndexVector.begin() + begin, h_faceIndexVector.begin() + end - 1);
				t0 = h_adjTriMap[temp][0];
				t1 = h_adjTriMap[temp][2];
				t2 = h_adjTriMap[temp][2];
			}
			h_seedElements.push_back(temp);
			begin += patchSize;

			count++;
		}
			
	}
	//allocate seed array.
	hipError_t status = hipMalloc(&d_seedArray, sizeof(int) * h_seedElements.size());
	if (status != hipSuccess)
	{
		std::cout << "allocation error for d_seedArray" << std::endl;
	}
	hipMemcpy(d_seedArray, h_seedElements.data(), sizeof(int)* h_seedElements.size(), hipMemcpyHostToDevice);
}

void preRxMeshDataStructure::h_initialiseSeedElementsMultiComp(TriangleMesh* tm, ComponentManager* cm)
{
	multiComponentPatchSize.clear();
	multiComponentPatchSize.resize(cm->componentCount);
	
	for (int i = 0; i < cm->componentCount; ++i)
	{
		int count{ 0 };
		int begin = cm->componentLocation[i], end = cm->componentLocation[i];
		int temp{ 0 };
		int start = cm->componentLocation[i];
		int stop = cm->componentLocation[i + 1];
		int faceCount = stop - start;
		multiComponentPatchSize[i] = (faceCount + multiComponentPatchCount[i] - 1) / multiComponentPatchCount[i];
		while (count < multiComponentPatchCount[i])
		{
			end += multiComponentPatchSize[i];
			if (end > numFaces)
				end = numFaces;

			int t0{ -1 }, t1{ -1 }, t2{-1};
			while (t0 == -1 && t1 == -1 && t2 == -1)
			{
				temp = *select_randomly(h_faceIndexVector.begin() + begin, h_faceIndexVector.begin() + end - 1);
				t0 = h_adjTriMap[temp][0];
				t1 = h_adjTriMap[temp][2];
				t2 = h_adjTriMap[temp][2];
			}
			h_seedElements.push_back(temp);
			begin += multiComponentPatchSize[i];

			count++;
		}
	}
}

void preRxMeshDataStructure::clear()
{
	h_seedElements.clear();
	h_adjascentTriangles.clear();
	multiComponentPatchCount.clear();
	multiComponentPatchSize.clear();
	h_adjTriMap.clear();
}

void preRxMeshDataStructure::clearSeedComponents(TriangleMesh* tm)
{
	h_seedElements.clear();
	multiComponentPatchSize.clear();
	int size_N = tm->faceVector.size() / 3;
	h_patchingArray.resize(size_N);
	std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);
}


void preRxMeshDataStructure::h_fillAdjascentTriangles(TriangleMesh* tm)
{
	clear();
	int size_N = tm->faceVector.size();
	int threadCount = 1 << 10;
	if (threadCount > size_N)
		threadCount = size_N;

	int gridSize = (size_N + threadCount - 1) / threadCount;
	h_adjascentTriangles.resize(size_N);
	int sharedMemorySize = 2 * threadCount;
	//in a manifold the max number of faces adjascent to one face is 3, but boundary vertices have -1 in this implementation.
	d_fillAdjascentTriangles << <gridSize, threadCount >> > (d_faceVector, d_adjascentTriangles, size_N);
	//copy the data for later operations.
	hipMemcpy(h_adjascentTriangles.data(), d_adjascentTriangles, sizeofFaceVector, hipMemcpyDeviceToHost);

	//fill hash map for faster access.
	for (int i = 0; i < h_adjascentTriangles.size(); i = i + 3)
	{
		h_adjTriMap[i / 3] = { h_adjascentTriangles[i], h_adjascentTriangles[i + 1], h_adjascentTriangles[i + 2] };
	}
}

__global__
void d_fillAdjascentTriangles(int* d_faceVector, int* d_adjascentTriangles, int size_N)
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	int lId = threadIdx.x;
	//basic modulo operation for triangles.

	if (tId < size_N)
	{
		int v0 = d_faceVector[tId];
		int v1 = 0;
		if (tId % 3 == 2)
			v1 = d_faceVector[tId - 2];
		else
			v1 = d_faceVector[tId + 1];
		int v2, v3;
		for (int i = 0; i < size_N; ++i)
		{
			v2 = d_faceVector[i];
			if(i % 3 == 2)
				v3 = d_faceVector[i - 2];
			else
				v3 = d_faceVector[i + 1];
			if (v0 == v3 && v1 == v2)
			{
				d_adjascentTriangles[tId] = i/3;
			}
		}

	}
}

__global__
void d_populatePatchingArray(int* d_patchingArray, int size_N, int* d_adjascentTriangles)
{
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	if (tId < size_N)
	{
		//the idea is to check for the faces who have adjascent elements in a different patch.
		//store that in boundary.
		//populate based on adj triangles. so no invalid triangle pops up in the patch.
		//thread divergence better than o(n3)

		if (d_patchingArray[tId] != -1)
		{
			int patch = d_patchingArray[tId];
			int t0 = d_adjascentTriangles[tId * 3];
			int t1 = d_adjascentTriangles[tId * 3 + 1];
			int t2 = d_adjascentTriangles[tId * 3 + 2];

			if (t0 != -1)
			{
				atomicCAS(d_patchingArray + t0, -1, patch);
				//printf("t0 %d \t %d \n ", t0, d_patchingArray[t0]);
			}
			if (t1 != -1)
			{
				atomicCAS(d_patchingArray + t1, -1, patch);
				//printf("t1 %d \t %d \n ", t1, d_patchingArray[t1]);
			}
			if (t2 != -1)
			{
				atomicCAS(d_patchingArray + t2, -1, patch);
				//printf("t2 %d \t %d \n ", t2, d_patchingArray[t2]);
			}

		}
	}
}


__global__
void d_counter(int* d_patchingArray, int size_N, int* d_count)
{
	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	*d_count = 0;
	if (tId < size_N)
	{
		if (d_patchingArray[tId] == -1)
			atomicAdd(d_count, 1);
	}
}

void preRxMeshDataStructure::h_fillPatchingArrayWithSeedPoints()
{
	//there is no point in parallelising this block.
	//and will only be done once.
	
	for (int i = 0; i < h_seedElements.size(); ++i)
	{
		int currFace = h_seedElements[i];
		h_patchingArray[currFace] = i;
	}

	hipError_t status = hipMemcpy(d_patchingArray, h_patchingArray.data(), sizeof(int) * h_patchingArray.size(), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		std::cout << "memcpy failed for d_patchingArray" << std::endl;
	}

	std::vector<int> test2(h_patchingArray.size(), -1);
	status = hipMemcpy(test2.data(), d_patchingArray, sizeof(int) * h_patchingArray.size(), hipMemcpyDeviceToHost); 

}

void preRxMeshDataStructure::h_populatePatches(TriangleMesh* tm, bool doIterations, ComponentManager* cm, int pc)
{

	//the algorithm involvest the following steps
	//initialise seed elements.
	//copy seed elements to patching array
	//for every non -1 element in patching array, add its neighbours.
	//keep counter to keep track of the patching process.
	//check if the faces are boundary. Select non boundary faces as seed for next itertaion.
	//repeat until 5th loop if iteration is enabled. 

	
	std::random_device rd;
	std::mt19937 gen(rd());


	clearSeedComponents(tm);
	h_initialiseSeedElements(tm, cm, pc);
	h_tempPatchArray.resize(h_patchingArray.size());
	std::fill(h_tempPatchArray.begin(), h_tempPatchArray.end(), -1);

	// i am putting 5 loops as convergence max.
	
	int loopCounter = 5;
	
	do{
		h_fillPatchingArrayWithSeedPoints();
		//clear the gpu values 

		int threadCount = patchSize;
		int blockCount = patchCount;
		int size_N = tm->faceVector.size() / 3;
		int sharedMemorySize = threadCount * sizeof(int);
		//set any random non zero value.
		int count = 0;
		int* d_count = 0;


		int* d_newPatchArray = 0;
		hipMalloc(&d_newPatchArray, sizeof(int) * h_patchingArray.size());
		
		std::vector<int> indidualCounter(patchCount, 0);
		std::vector<int> prefixSum(patchCount, 0);
		std::vector<int> h_newPatchingArray(h_patchingArray.size(), 0);


		int* d_individualCounts = 0;
		hipMalloc(&d_individualCounts, sizeof(int) * patchCount);
		hipMemcpy(d_individualCounts, indidualCounter.data(), sizeof(int)* patchCount, hipMemcpyHostToDevice);


		int* d_prefixSum = 0;
		hipMalloc(&d_prefixSum, sizeof(int) * patchCount);

		hipMalloc(&d_count, sizeof(int));
		hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
		//i was using blelloch earlier to get the sum of all face values.
		//but the reference in nvidea is for a single block.
		/*for (int i = 0; i < 3; ++i)*/

		
		do
		{
			hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
			d_populatePatchingArray << <blockCount, threadCount >> > (d_patchingArray, size_N, d_adjascentTriangles);
			hipDeviceSynchronize();
			
			d_counter << <blockCount, threadCount >> > (d_patchingArray, size_N, d_count);
			hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
		} while (count != 0);

		hipDeviceSynchronize();

	

		//grouping code
		//blelloch code given in nvidea works only for one block.
		
		d_computePatchCount << <blockCount, threadCount >> > (d_patchingArray, d_individualCounts, h_patchingArray.size());
		

		hipMemcpy(indidualCounter.data(), d_individualCounts, patchCount * sizeof(int), hipMemcpyDeviceToHost);

		prefixSum[0] = 0;
		for (int i = 1; i < patchCount; ++i) {
			prefixSum[i] = prefixSum[i - 1] + indidualCounter[i - 1];
		}

		hipMemcpy(d_prefixSum, prefixSum.data(), sizeof(int) * patchCount, hipMemcpyHostToDevice);
		hipMemset(d_individualCounts, 0, patchCount * sizeof(int));

		d_arrangePatches << <blockCount, threadCount >> > (d_patchingArray, d_newPatchArray, d_individualCounts, d_prefixSum, h_patchingArray.size());

		hipMemcpy(h_newPatchingArray.data(), d_newPatchArray, sizeof(int) * size_N, hipMemcpyDeviceToHost);
		h_seedElements.clear();
		//select new seed points.
		int begin{ 0 };
		int end{ 0 };
		int temp{ 0 };

		//do this only for the last loop
		if(loopCounter == 1)
			hipMemcpy(h_tempPatchArray.data(), d_patchingArray, sizeof(int) * size_N, hipMemcpyDeviceToHost);

		for (int i = 0; i < h_newPatchingArray.size(); ++i)
		{
		//update seed step.
		// basically you update the seed points to make it centralised.	
			if (begin < patchSize)
			{
				begin = i * patchSize;
				end = i * patchSize + patchSize;

				int t0{ -1 }, t1{ -1 }, t2{ -1 };
				bool breakLoop = false;
				//safety.
				int count = 50;
				while (count && !breakLoop)
				{

					temp = *select_randomly(h_newPatchingArray.begin() + begin, h_newPatchingArray.begin() + end - 1);
					t0 = h_adjTriMap[temp][0];
					t1 = h_adjTriMap[temp][1];
					t2 = h_adjTriMap[temp][2];

					if (h_tempPatchArray[t0] == i && h_tempPatchArray[t1] == i && h_tempPatchArray[t2] == i)
					{
						breakLoop = true;
					}
					count--;
				}
				h_seedElements.push_back(temp);
			}
		}
		
		hipMemset(d_patchingArray, -1, size_N * sizeof(int));
		hipMemcpy(d_patchingArray, h_newPatchingArray.data(), sizeof(int) * h_newPatchingArray.size(), hipMemcpyHostToDevice);
		std::fill(h_patchingArray.begin(), h_patchingArray.end(), -1);
	
	
		loopCounter--;
		hipFree(d_prefixSum);
		hipFree(d_newPatchArray);
		hipFree(d_individualCounts);

	}while (loopCounter);

	hipMemcpy(h_patchingArray.data(), d_patchingArray, sizeof(int)* h_patchingArray.size(), hipMemcpyDeviceToHost);
	
}


void preRxMeshDataStructure::addRibbons(TriangleMesh* tm)
{
	int size_N = tm->faceVector.size() / 3;
	
	std::vector<int> NegData(size_N, -1);

	hipMemcpy(d_boundaryElements, NegData.data(), sizeof(int) * size_N, hipMemcpyHostToDevice);

	int threadCount = patchSize;
	int blockCount = patchCount;

	hipMalloc(&d_patchPositions, size_N * sizeof(int));
	hipMemcpy(d_patchPositions, h_tempPatchArray.data(), size_N * sizeof(int), hipMemcpyHostToDevice);

	d_findBoundaryPoints << <blockCount, threadCount >> > (d_patchingArray, size_N, d_boundaryElements, d_adjascentTriangles, d_patchPositions);
	hipMemcpy(h_boundaryElements.data(), d_boundaryElements, sizeof(int) * size_N, hipMemcpyDeviceToHost);

	int a = 30;
}


__global__ void d_computePatchCount(int* d_patchingArray, int* d_individualCounts, int size_N) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < size_N) {
		int value = d_patchingArray[tId];
		atomicAdd(&d_individualCounts[value], 1);
	}
}

__global__ void d_arrangePatches(int* d_patchingArray, int* d_newPatchArray, int* d_individualCounts, int* d_prefixSum, int size_N) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < size_N) {
		int value = d_patchingArray[tId];
		//printf("%d \n", value);
		int pos = atomicAdd(&d_individualCounts[value], 1);
		d_newPatchArray[d_prefixSum[value] + pos] = tId;
	}
}


__global__
void d_findBoundaryPoints(int* d_patchingArray, int size_N, int* d_boundaryElements, int* d_adjascentTriangles, int* d_patchPositions)
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < size_N)
	{
		int current = d_patchingArray[tId];
		if (current != -1)
		{
			int begin = blockIdx.x * blockDim.x;
			int end = blockIdx.x * blockDim.x + blockDim.x;
			int t0{ -1 }, t1{ -1 }, t2{ -1 };
			t0 = d_adjascentTriangles[3 * current];
			t1 = d_adjascentTriangles[3 * current + 1];
			t2 = d_adjascentTriangles[3 * current + 2];
			if (t0 != -1 && d_patchPositions[t0] != blockIdx.x)
				d_boundaryElements[current] = blockIdx.x;
			if (t1 != -1 && d_patchPositions[t1] != blockIdx.x)
				d_boundaryElements[current] = blockIdx.x;
			if (t2 != -1 && d_patchPositions[t2] != blockIdx.x)
				d_boundaryElements[current] = blockIdx.x;
		}
		
	}

}